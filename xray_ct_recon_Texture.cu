#include "hip/hip_runtime.h"

/* 
Based off work by Nelson, et al.
Brigham Young University (2010)

Adapted by Kevin Yuh (2015)

Modified by Jordan Bonilla and Matthew Cedeno (2016)
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include "ta_utilities.hpp"

#define PI 3.14159265358979




/* Check errors on CUDA runtime functions */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

// declare texture reference 
texture<float, 2, hipReadModeElementType> texreference;

/* Check errors on cuFFT functions */
void gpuFFTchk(int errval){
    if (errval != HIPFFT_SUCCESS){
        printf("Failed FFT call, error code %d\n", errval);
    }
}


/* Check errors on CUDA kernel calls */
void checkCUDAKernelError()
{
    hipError_t err = hipGetLastError();
    if  (hipSuccess != err){
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    } else {
        fprintf(stderr, "No kernel error detected\n");
    }

}

__global__ 
void cudaMultiplyKernel(hipfftComplex *raw_data, 
                        unsigned int nAngles, unsigned int sinogram_width) {
    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    while (thread_index < nAngles * sinogram_width) {
        unsigned int p = thread_index % sinogram_width; 
        if (p < sinogram_width / 2) {
            raw_data[thread_index].x = raw_data[thread_index].x * ((2.0 * p) / sinogram_width);
            raw_data[thread_index].y = raw_data[thread_index].y * ((2.0 * p) / sinogram_width);
        } else {
            raw_data[thread_index].x = raw_data[thread_index].x * ((2.0 * (sinogram_width - p)) / sinogram_width);
            raw_data[thread_index].y = raw_data[thread_index].y * ((2.0 * (sinogram_width - p)) / sinogram_width);

        }
        
        // raw_data[thread_index].x = raw_data[thread_index].x;
        
        // raw_data[thread_index].y = raw_data[thread_index].y;
        thread_index += blockDim.x * gridDim.x;
    }
}

__global__
void cudaTakeFloatKernel(const hipfftComplex *dev_out_filter, 
                        float *dev_sinogram_float, const unsigned int nAngles, const unsigned int sinogram_width) {
    unsigned int thread_index = blockDim.x * blockIdx.x + threadIdx.x;
    while (thread_index < nAngles * sinogram_width) {
        dev_sinogram_float[thread_index] = dev_out_filter[thread_index].x;
        thread_index += blockDim.x * gridDim.x;
    }
}


__global__
void cudaBackProjKernel(float *output_dev, 
                        const unsigned int nAngles, 
                        const unsigned int sinogram_width,
                        const unsigned int width, 
                        const unsigned int height) {

    unsigned int thread_index = blockDim.x * blockIdx.x + threadIdx.x;

    while (thread_index < width * height) {
        int y0 = height/2 - thread_index / width;
        int x0 = thread_index % width;

        for (int i = 0; i < nAngles; ++i) {
            float sita = (float)i * PI / nAngles;
            float d, xi, yi, q;
            if (sita == 0) {
                d = x0;
            } else if (sita == PI / 2) {
                d = y0;
            } else {
                float m = -cos(sita)/sin(sita);
                q = -1/m;
                xi = (y0 - m * x0)/(q - m);
                yi = q * xi;
                d = sqrtf(xi * xi + yi * yi);
            }
            if ((q > 0 && xi < 0)||(q < 0 && xi > 0)) {
                output_dev[thread_index] += tex2D(texreference, sinogram_width/2-d, i); // ( , xindex, yindex)
            } else {
                output_dev[thread_index] += tex2D(texreference, sinogram_width/2+d, i);
                // output_dev[index] += dev_sinogram_float[(int)(i * sinogram_width + d + sinogram_width / 2)];
            }
        }
        thread_index += blockDim.x * gridDim.x;
    }

}



void cudaCallMultiplyKernel (const unsigned int blocks, 
                            const unsigned int threadsPerBlock,
                            hipfftComplex *raw_data,
                            const unsigned int nAngles, 
                            const unsigned int sinogram_width) {
    cudaMultiplyKernel<<<blocks, threadsPerBlock>>>(raw_data, nAngles, sinogram_width);
}


void cudaCallTakeFloatKernel(const unsigned int nBlocks, 
                            const unsigned int threadsPerBlock, 
                            const hipfftComplex *dev_out_filter, 
                            float *dev_sinogram_float, 
                            const unsigned int nAngles, 
                            const unsigned int sinogram_width) {
    cudaTakeFloatKernel<<<nBlocks, threadsPerBlock>>>(dev_out_filter, dev_sinogram_float, nAngles, sinogram_width);
}

void cudaCallBackProjKernel(const unsigned int nBlocks, 
                            const unsigned int threadsPerBlock, 
                            float *output_dev, 
                            const unsigned int nAngles, 
                            const unsigned int sinogram_width,
                            const unsigned int width, 
                            const unsigned int height) {
    cudaBackProjKernel<<<nBlocks, threadsPerBlock>>>(output_dev, nAngles, sinogram_width, width, height);
}




int main(int argc, char** argv){
    // These functions allow you to select the least utilized GPU
    // on your system as well as enforce a time limit on program execution.
    // Please leave these enabled as a courtesy to your fellow classmates
    // if you are using a shared computer. You may ignore or remove these
    // functions if you are running on your local machine.
    TA_Utilities::select_least_utilized_GPU();
    int max_time_allowed_in_seconds = 30;
    TA_Utilities::enforce_time_limit(max_time_allowed_in_seconds);

    // Begin timer and check for the correct number of inputs
    time_t start = clock();
    if (argc != 7){
        fprintf(stderr, "Incorrect number of arguments.\n\n");
        fprintf(stderr, "\nArguments: \n \
        < Input sinogram text file's name > \n \
        < Width or height of original image, whichever is larger > \n \
        < Number of angles in sinogram >\n \
        < threads per block >\n \
        < number of blocks >\n \
        < output text file's name >\n");
        exit(EXIT_FAILURE);
    }






    /********** Parameters **********/

    int width = atoi(argv[2]);
    int height = width;
    int sinogram_width = (int)ceilf( height * sqrt(2) );

    int nAngles = atoi(argv[3]);


    int threadsPerBlock = atoi(argv[4]);
    int nBlocks = atoi(argv[5]);


    /********** Data storage *********/


    // GPU DATA STORAGE
    hipfftComplex *dev_sinogram_cmplx;
    float *dev_sinogram_float; 
    float* output_dev;  // Image storage


    hipfftComplex *sinogram_host;

    size_t size_result = width*height*sizeof(float);
    float *output_host = (float *)malloc(size_result);




    /*********** Set up IO, Read in data ************/

    sinogram_host = (hipfftComplex *)malloc(  sinogram_width*nAngles*sizeof(hipfftComplex) );

    FILE *dataFile = fopen(argv[1],"r");
    if (dataFile == NULL){
        fprintf(stderr, "Sinogram file missing\n");
        exit(EXIT_FAILURE);
    }

    FILE *outputFile = fopen(argv[6], "w");
    if (outputFile == NULL){
        fprintf(stderr, "Output file cannot be written\n");
        exit(EXIT_FAILURE);
    }

    int j, i;

    for(i = 0; i < nAngles * sinogram_width; i++){
        fscanf(dataFile,"%f",&sinogram_host[i].x);
        sinogram_host[i].y = 0;
    }

    fclose(dataFile);


    /*********** Assignment starts here *********/

    /* TODO ok: Allocate memory for all GPU storage above, copy input sinogram
    over to dev_sinogram_cmplx. */
    // in texture memory:

    gpuErrchk(hipMalloc((void**)&dev_sinogram_cmplx, nAngles * sinogram_width * sizeof(hipfftComplex)));
    gpuErrchk(hipMemcpy(dev_sinogram_cmplx, sinogram_host, nAngles * sinogram_width * sizeof(hipfftComplex), hipMemcpyHostToDevice));



    /* TODO 1 ok: Implement the high-pass filter:
        - Use cuFFT for the forward FFT
        - Create your own kernel for the frequency scaling.
        - Use cuFFT for the inverse FFT
        - extract real components to floats
        - Free the original sinogram (dev_sinogram_cmplx)

        Note: If you want to deal with real-to-complex and complex-to-real
        transforms in cuFFT, you'll have to slightly change our code above.
    */

    // // create the high pass filter vector

    // hipfftComplex *filter_v = (hipfftComplex*)malloc(sizeof(hipfftComplex) * sinogram_width);
    // for (int i = 0; i < sinogram_width; ++i) {
    //     filter_v[i].x = 1 - abs((float)(2 * i - sinogram_width) / sinogram_width);
    //     filter_v[i].y = 0;
    // } // on freq domain

    // DATA storage
    // hipfftComplex *dev_filter_v;
    // gpuErrchk(hipMalloc((void**)&dev_filter_v, sizeof(hipfftComplex) * sinogram_width));
    // gpuErrchk(hipMemcpy(dev_filter_v, filter_v, sinogram_width * sizeof(hipfftComplex), hipMemcpyHostToDevice));
    // hipfftComplex *dev_out_filter;
    // gpuErrchk(hipMalloc((void**)&dev_out_filter, sizeof(hipfftComplex) * sinogram_width * nAngles));

    hipfftHandle plan;
    gpuFFTchk(hipfftPlan1d(&plan, sinogram_width, HIPFFT_C2C, nAngles));
    gpuFFTchk(hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_FORWARD));

    // call the kernel to perform the filter
    cudaCallMultiplyKernel(nBlocks, threadsPerBlock, dev_sinogram_cmplx, nAngles, sinogram_width);
    checkCUDAKernelError();
    printf("finish filter\n");
    // inverse fft
    gpuFFTchk(hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_BACKWARD));
    // destroy the cufft plan
    gpuFFTchk(hipfftDestroy(plan));

    // take the float
    gpuErrchk(hipMalloc((void**)&dev_sinogram_float, nAngles * sinogram_width * sizeof(float)));
    cudaCallTakeFloatKernel(nBlocks, threadsPerBlock, dev_sinogram_cmplx, dev_sinogram_float, nAngles, sinogram_width);
    checkCUDAKernelError();
    // free dev_sinogram_cmplx
    gpuErrchk(hipFree(dev_sinogram_cmplx));
    printf("finish fft\n");
    // gpuErrchk(hipFree(dev_out_filter));

    /* TODO 2: Implement backprojection.
        - Allocate memory for the output image.
        - Create your own kernel to accelerate backprojection.
        - Copy the reconstructed image back to output_host.
        - Free all remaining memory on the GPU.
    */

    // first I think I will have to copy dev_sinogram_float from device to host;
    // but actually it's not necessary
    // float *host_sinogram_float = (float*)malloc(sizeof(float)*nAngles*sinogram_width);
    // gpuErrchk(hipMemcpy(host_sinogram_float, dev_sinogram_float, sizeof(float)*nAngles*sinogram_width, hipMemcpyDeviceToHost));
    // gpuErrchk(hipFree(dev_sinogram_float));

    hipArray *cArray;
    hipChannelFormatDesc channel;

    channel = hipCreateChannelDesc<float>();

    // allocate device memory for cuda array
    gpuErrchk(hipMallocArray(&cArray, &channel, nAngles, sinogram_width));
    int bytes = sizeof(float) * nAngles * sinogram_width;
    gpuErrchk(hipMemcpyToArray(cArray, 0, 0, dev_sinogram_float, bytes, hipMemcpyDeviceToDevice)); 
    // copy float sinogram from gloabl memory to texture memory =(
    gpuErrchk(hipFree(dev_sinogram_float));

    // set texture filter mode
    texreference.filterMode = hipFilterModeLinear;

    // set texture address mode
    texreference.addressMode[0] = hipAddressModeWrap; // necessary???
    texreference.addressMode[1] = hipAddressModeClamp;

    // bind texture reference with cuda array
    gpuErrchk(hipBindTextureToArray(texreference, cArray));

    // Allocate memory for the output image.
    gpuErrchk(hipMalloc((void**)&output_dev, size_result));
    gpuErrchk(hipMemset(output_dev, 0, size_result));

    // call kernel
    cudaCallBackProjKernel(nBlocks, threadsPerBlock, output_dev, nAngles, sinogram_width, width, height);
    checkCUDAKernelError();

    // unbind texture
    hipUnbindTexture(texreference);

    // copy result out
    gpuErrchk(hipMemcpy(output_host, output_dev, size_result, hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(output_dev));
    gpuErrchk(hipFreeArray(cArray));



    // Allocate memory for the output image.
    // gpuErrchk(hipMalloc((void**)&output_dev, size_result));
    // gpuErrchk(hipMemset(output_dev, 0, size_result));
    // // call back projection kernel
    // cudaCallBackProjKernel(nBlocks, threadsPerBlock, dev_sinogram_float, output_dev, nAngles, sinogram_width, width, height);
    // printf("finish back proj\n");
    // checkCUDAKernelError();
    // gpuErrchk(hipMemcpy(output_host, output_dev, size_result, hipMemcpyDeviceToHost));
    // gpuErrchk(hipFree(dev_sinogram_float));
    // gpuErrchk(hipFree(output_dev));


    /* Export image data. */

    for(j = 0; j < width; j++){
        for(i = 0; i < height; i++){
            fprintf(outputFile, "%e ",output_host[j*width + i]);
        }
        fprintf(outputFile, "\n");
    }


    /* Cleanup: Free host memory, close files. */

    free(sinogram_host);
    free(output_host);

    fclose(outputFile);
    printf("CT reconstruction complete. Total run time: %f seconds\n", (float) (clock() - start) / 1000.0);
    return 0;
}



